#include "hip/hip_runtime.h"
//   Copyright 2022 <Huawei Technologies Co., Ltd>
//
//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at
//
//       http://www.apache.org/licenses/LICENSE-2.0
//
//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.
#include <thrust/transform_reduce.h>

#include "config/openmp.hpp"

#include "simulator/utils.hpp"
#include "simulator/vector/detail/gpu_vector_double_policy.cuh"
#include "simulator/vector/detail/gpu_vector_float_policy.cuh"
#include "simulator/vector/detail/gpu_vector_policy.cuh"
#include "thrust/device_ptr.h"
#include "thrust/functional.h"
#include "thrust/inner_product.h"

namespace mindquantum::sim::vector::detail {

template <typename derived_, typename calc_type_>
template <index_t mask, index_t condi, class binary_op>
void GPUVectorPolicyBase<derived_, calc_type_>::ConditionalBinary(qs_data_p_t src, qs_data_p_t des,
                                                                  qs_data_t succ_coeff, qs_data_t fail_coeff,
                                                                  index_t dim, const binary_op& op) {
    thrust::counting_iterator<size_t> i(0);
    thrust::for_each(i, i + dim, [=] __device__(size_t i) {
        if ((i & mask) == condi) {
            des[i] = op(src[i], succ_coeff);
        } else {
            des[i] = op(src[i], fail_coeff);
        }
    });
}

template <typename derived_, typename calc_type_>
template <class binary_op>
void GPUVectorPolicyBase<derived_, calc_type_>::ConditionalBinary(qs_data_p_t src, qs_data_p_t des, index_t mask,
                                                                  index_t condi, qs_data_t succ_coeff,
                                                                  qs_data_t fail_coeff, index_t dim,
                                                                  const binary_op& op) {
    thrust::counting_iterator<size_t> i(0);
    thrust::for_each(i, i + dim, [=] __device__(size_t i) {
        if ((i & mask) == condi) {
            des[i] = op(src[i], succ_coeff);
        } else {
            des[i] = op(src[i], fail_coeff);
        }
    });
}

template <typename derived_, typename calc_type_>
void GPUVectorPolicyBase<derived_, calc_type_>::ConditionalAdd(qs_data_p_t src, qs_data_p_t des, index_t mask,
                                                               index_t condi, qs_data_t succ_coeff,
                                                               qs_data_t fail_coeff, index_t dim) {
    derived::template ConditionalBinary(src, des, mask, condi, succ_coeff, fail_coeff, dim, thrust::plus<qs_data_t>());
}

template <typename derived_, typename calc_type_>
void GPUVectorPolicyBase<derived_, calc_type_>::ConditionalMinus(qs_data_p_t src, qs_data_p_t des, index_t mask,
                                                                 index_t condi, qs_data_t succ_coeff,
                                                                 qs_data_t fail_coeff, index_t dim) {
    derived::template ConditionalBinary(src, des, mask, condi, succ_coeff, fail_coeff, dim, thrust::minus<qs_data_t>());
}

template <typename derived_, typename calc_type_>
void GPUVectorPolicyBase<derived_, calc_type_>::ConditionalMul(qs_data_p_t src, qs_data_p_t des, index_t mask,
                                                               index_t condi, qs_data_t succ_coeff,
                                                               qs_data_t fail_coeff, index_t dim) {
    derived::template ConditionalBinary(src, des, mask, condi, succ_coeff, fail_coeff, dim,
                                        thrust::multiplies<qs_data_t>());
}

template <typename derived_, typename calc_type_>
void GPUVectorPolicyBase<derived_, calc_type_>::ConditionalDiv(qs_data_p_t src, qs_data_p_t des, index_t mask,
                                                               index_t condi, qs_data_t succ_coeff,
                                                               qs_data_t fail_coeff, index_t dim) {
    derived::template ConditionalBinary(src, des, mask, condi, succ_coeff, fail_coeff, dim,
                                        thrust::divides<qs_data_t>());
}

template <typename derived_, typename calc_type_>
void GPUVectorPolicyBase<derived_, calc_type_>::QSMulValue(qs_data_p_t src, qs_data_p_t des, qs_data_t value,
                                                           index_t dim) {
    derived::template ConditionalBinary<0, 0>(src, des, value, 0, dim, thrust::multiplies<qs_data_t>());
}

template <typename derived_, typename calc_type_>
auto GPUVectorPolicyBase<derived_, calc_type_>::ConditionalCollect(qs_data_p_t qs, index_t mask, index_t condi,
                                                                   bool abs, index_t dim) -> qs_data_t {
    qs_data_t res = 0;
    thrust::counting_iterator<size_t> l(0);
    if (abs) {
        res = thrust::transform_reduce(
            l, l + dim,
            [=] __device__(size_t l) {
                if ((l & mask) == condi) {
                    return thrust::conj(qs[l]) * qs[l];
                }
                return qs_data_t(0.0, 0.0);
            },
            qs_data_t(0, 0), thrust::plus<qs_data_t>());
    } else {
        res = thrust::transform_reduce(
            l, l + dim,
            [=] __device__(size_t l) {
                if ((l & mask) == condi) {
                    return thrust::conj(qs[l]) * qs[l];
                }
                return qs_data_t(0.0, 0.0);
            },
            qs_data_t(0, 0), thrust::plus<qs_data_t>());
    }
    return res;
}

template struct GPUVectorPolicyBase<GPUVectorPolicyFloat, float>;
template struct GPUVectorPolicyBase<GPUVectorPolicyDouble, double>;

}  // namespace mindquantum::sim::vector::detail
